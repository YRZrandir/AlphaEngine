#include "hip/hip_runtime.h"
#include "CudaMatrixHelpers.cuh"
#include <iostream>

namespace
{
hipsparseHandle_t sCuSparseContext = nullptr;
void CheckCusparseInit()
{
    if (sCuSparseContext == nullptr)
    {
        hipsparseCreate( &sCuSparseContext );
    }
}

}
__global__ void CUDAvplusv_impl( float* a, float sa, float* b, float sb, float* dst, int size );

hipsparseSpMatDescr_t CreateCUDASparseMatrix( const Eigen::SparseMatrix<float>& m )
{
    CheckCusparseInit();
    if (!m.isCompressed())
    {
        std::cout << "ERROR: m has to be compressed." << std::endl;
        return nullptr;
    }

    int* dA_cscOffsets, * dA_rows;
    float* dA_values;

    int A_num_rows = m.rows();
    int A_num_cols = m.cols();
    int A_nnz = m.nonZeros();
    hipMalloc( (void**)&dA_cscOffsets, (A_num_cols + 1) * sizeof( int ) );
    hipMalloc( (void**)&dA_rows, A_nnz * sizeof( int ) );
    hipMalloc( (void**)&dA_values, A_nnz * sizeof( float ) );

    hipMemcpy( dA_cscOffsets, m.outerIndexPtr(), (A_num_cols + 1) * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( dA_rows, m.innerIndexPtr(), A_nnz * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( dA_values, m.valuePtr(), A_nnz * sizeof( float ), hipMemcpyHostToDevice );

    hipsparseSpMatDescr_t sparse_mat;
    hipsparseStatus_t status = hipsparseCreateCsc( &sparse_mat, A_num_rows, A_num_cols, A_nnz, dA_cscOffsets, dA_rows, dA_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F );
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "ERROR: hipsparseCreateCsc Failed." << std::endl;
    }
    return sparse_mat;
}

hipsparseDnVecDescr_t CreateCUDAVector( const CudaBuffer<float>& vec )
{
    CheckCusparseInit();
    hipsparseDnVecDescr_t cu_vec;
    hipsparseCreateDnVec( &cu_vec, vec.Count(), (void*)vec.Data(), HIP_R_32F );
    return cu_vec;
}

void SetCUDAVector( hipsparseDnVecDescr_t cu_vec, const CudaBuffer<float>& mem )
{
    CheckCusparseInit();
    hipsparseDnVecSetValues( cu_vec, (void*)mem.Data() );
}

void ReadCUDAVector( hipsparseDnVecDescr_t cu_vec, CudaBuffer<float>& mem )
{
    CheckCusparseInit();
    hipsparseDnVecGetValues( cu_vec, (void**)&mem.Data() );
}

size_t CUDASpmvBufferSize( hipsparseSpMatDescr_t M, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t y )
{
    float alpha = 1.0f;
    float beta = 0.0f;
    size_t buffer_size = 0;
    hipsparseSpMV_bufferSize( sCuSparseContext, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, M, x, &beta, y, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &buffer_size );
    return buffer_size;
}

void CUDASpmv( hipsparseSpMatDescr_t M, hipsparseDnVecDescr_t x, hipsparseDnVecDescr_t y, void* buffer )
{
    float alpha = 1.0f;
    float beta = 0.0f;
    bool need_free = false;
    if (buffer == nullptr)
    {
        need_free = true;
        size_t buffer_size = 0;
        hipsparseSpMV_bufferSize( sCuSparseContext, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, M, x, &beta, y, HIP_R_32F,
            HIPSPARSE_MV_ALG_DEFAULT, &buffer_size );
        hipMalloc( &buffer, buffer_size );
    }
    hipsparseSpMV( sCuSparseContext, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, M, x, &beta, y, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, buffer );
    if (need_free)
    {
        hipFree( buffer );
    }
}

void CUDAvplusv( CudaBuffer<float>& a, float sa, CudaBuffer<float>& b, float sb, CudaBuffer<float>& dst, dim3 gridsize, dim3 blocksize )
{
    if (!(a.Count() == b.Count() && a.Count() == dst.Count()))
    {
        __debugbreak();
    }
    CUDAvplusv_impl << <gridsize, blocksize >> > (a.Data(), sa, b.Data(), sb, dst.Data(), a.Count());
}

__global__ void CUDAvplusv_impl( float* a, float sa, float* b, float sb, float* dst, int size )
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    dst[i] = a[i] * sa + b[i] * sb;
}
